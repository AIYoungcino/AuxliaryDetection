#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <ctime>
#include <iostream>
#define BLOCK_SIZE 256
__global__ void gpu_matrix_mult(int* a, int* b, int* c, int m, int n, int k)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int sum = 0;
	if (col < k && row < m)
	{
		for (int i = 0; i < n; i++)
		{
			sum += a[row * n + i] * b[i * k + col];
		}
		c[row * k + col] = sum;
	}
}

int main()
{
	int m=1024, n=1024, k=1024;
	srand(time(NULL));
	int* h_a, * h_b, * h_c, * h_cc;
	hipHostMalloc((void**)&h_a, sizeof(int) * m * n);
	hipHostMalloc((void**)&h_b, sizeof(int) * n * k);
	hipHostMalloc((void**)&h_c, sizeof(int) * m * k);
	hipHostMalloc((void**)&h_cc, sizeof(int) * m * k);
	for (int i = 0; i < m; ++i) {
		for (int j = 0; j < n; ++j) {
			h_a[i * n + j] = rand() % 1024;
		}
	}
	for (int i = 0; i < n; ++i) {
		for (int j = 0; j < k; ++j) {
			h_b[i * k + j] = rand() % 1024;
		}
	}
	float gpu_elapsed_time_ms;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	int* d_a, * d_b, * d_c;
	hipMalloc((void**)&d_a, sizeof(int) * m * n);
	hipMalloc((void**)&d_b, sizeof(int) * n * k);
	hipMalloc((void**)&d_c, sizeof(int) * m * k);
	hipMemcpy(d_a, h_a, sizeof(int) * m * n, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, sizeof(int) * n * k, hipMemcpyHostToDevice);
	unsigned int grid_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
	unsigned int grid_cols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
	dim3 dimGrid(grid_cols, grid_rows);
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	gpu_matrix_mult << <dimGrid, dimBlock >> > (d_a, d_b, d_c, m, n, k);
	hipMemcpy(h_c, d_c, sizeof(int) * m * k, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop);
	printf("Time elapsed on matrix multiplication of %dx%d.\n%dx%d on GPU: %f ms.\n\n", m, n, n, k, gpu_elapsed_time_ms);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	hipHostFree(h_a);
	hipHostFree(h_b);
	hipHostFree(h_c);
	hipHostFree(h_cc);
	std::cin.get();
	return 0;
}
